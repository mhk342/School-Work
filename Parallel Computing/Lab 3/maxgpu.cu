#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 1024
#define WARP 32

long getmax(long *, long);
__global__ void get_block_max(long arr[], int size, int thread_size);

int main(int argc, char *argv[])
{
   long size = 0;  // The size of the array
   long i;  // loop index
   long * numbers; //pointer to the array
   long * array;
   long max = 0;
   
	hipSetDevice(1);
    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }

    size = atol(argv[1]);
    
	


    numbers = (long *)malloc(size * sizeof(long));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %ld\n", size);
       exit(1);
    }
	

	
    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1
    for( i = 0; i < size; i++) {
       numbers[i] = rand() % size;
	}
	
	int elements = size - (size % WARP);
	int threads = (int)ceil((double)elements/BLOCK_SIZE);
	int blocks = (int)ceil(((double)threads/BLOCK_SIZE));
	
	hipError_t err = hipMalloc((void**)&array, sizeof(long) * elements);
	if (err != hipSuccess) {
		printf("hipMalloc failure\n");
	}
	err = hipMemcpy(array, numbers, sizeof(long) * elements, hipMemcpyHostToDevice);

	
	get_block_max<<<blocks, BLOCK_SIZE>>>(array, elements, BLOCK_SIZE);
	
	
	for (i = elements; i < size; i++) {
		if (max < numbers[i]) {
			max = numbers[i];
		}
	}
	
	hipMemcpy(numbers, array, sizeof(long) * blocks, hipMemcpyDeviceToHost);
	
	for (i = 0; i < blocks; i++) {
		if (numbers[i] > max) {
			max = numbers[i];
		}
	}

	
	
    printf(" The maximum number in the array is: %ld\n", max);

	hipFree(array);
    free(numbers);
    exit(0);
}

// method that finds max within a block
__global__
void get_block_max(long arr[], int size, int thread_size) {
	int i;
	int max = 0;
	__shared__ int sdata[BLOCK_SIZE];
	
	unsigned int index = (threadIdx.x + blockIdx.x * blockDim.x) * thread_size;

	for (i = index; i < index + thread_size && i < size; i++) {
		if (max < arr[i]) {
			max = arr[i];
		}
	}
	sdata[threadIdx.x] = max;
	__syncthreads();

	unsigned int thid = threadIdx.x;
	unsigned int stride;

	for (stride = blockDim.x/2; stride >= WARP; stride >>= 1) {
		if (thid < stride && thid + stride < blockDim.x) {
			int curr = sdata[thid + stride];
			if (sdata[thid] < curr) {
				sdata[thid] = curr;
			}
		}
		__syncthreads();
	} 

	__syncthreads();

	if (thid == 0) {
		int i;
		int max = 0;

		for (i = thid; i < thid + WARP && i < size; i++) {
			if (max < sdata[i]) {
				max = sdata[i];
			}
		}
		sdata[0] = max;
		arr[blockIdx.x] = sdata[0];
	}
}




/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/
long getmax(long num[], long size)
{
  long i;
  long max = num[0];

  for(i = 1; i < size; i++)
	if(num[i] > max)
	   max = num[i];

  return( max );

}
